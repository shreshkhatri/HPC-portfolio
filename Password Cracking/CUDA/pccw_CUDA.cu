#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include<stdbool.h>

#define password_width 6
#define password_number 4

/****************************************************************************
  Compile and run with:
    nvcc -o  pccw_CUDA_object pccw_CUDA.cu 
    ./pccw_CUDA_object
  
***************************************************************************/


/* The passwords have been specified as array variable on device. This approach has elminated need for 
using CUDA library calls such as hipMalloc(),hipMemcpy() etc. Futhermore, specifying passwords as array has 
prevented those passwords being coppied to all the threads created since the array created has global scope*/

__device__ char passwordlist_device[password_number][password_width+1]={"DV7611","ET6122","HR4633","SD6744"};



int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


/* This function runs on device this function checks a combination of password against predefined password array.
This function returns true for password match and false otherwise. */
__device__ bool crack_password(char *input) {
char *cursor_copy;
cursor_copy=input;

for(int i=0;i<4;i++){
   for(int o=0;o<7;o++){
	if (passwordlist_device[i][o]==(char)*input){
		if(passwordlist_device[i][o]=='\0')
                {
                  return true;
                }
                input++;
          }        
	else
            break;	

   }
  
   //reseting of input pointer to the beginning for next iteration
   input=cursor_copy;

}

return false;

 
}


/****************************************************************************
This kernel function is called 26*26 times by 26*26 threads. infact 1 block 
contains 26 threads and there are 26 such blocks. The threadIdx.x and blockIdx.x 
dimensions have been used to create combinnation of two uppercase initials AA through to 
ZZ. The four nested FOR loops have been used to create combinations of digits ranging 
from 0000 to 9999. the array 'combination_pswd' is used to hold such combinations
one at a time. 
*****************************************************************************/

__global__ void  kernel_function() {
char k,l,m,n;
  
  char combination_pswd[7];
  combination_pswd[6] = '\0';
    
combination_pswd[0] = (char) blockIdx.x+65;
combination_pswd[1] = (char) threadIdx.x+65;
	for(k='0'; k<='9'; k++){
	  for(l='0'; l<='9'; l++){
             for(m='0'; m<='9'; m++){
                 for(n='0'; n<='9'; n++){
                 combination_pswd[2]=k;
		 combination_pswd[3]=l;
		 combination_pswd[4]=m;
		 combination_pswd[5]=n;
                  
                 if(crack_password(combination_pswd))
                    printf("Match found: %s\n",combination_pswd);

                 }

             }
	        
	}
	
    }

}


int main() {

  struct  timespec start_time, finish_time;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start_time);

  kernel_function<<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish_time);
  time_difference(&start_time, &finish_time, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}


